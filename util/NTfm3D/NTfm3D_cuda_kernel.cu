#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <typeinfo>


// #if __CUDA_ARCH__ < 600
// __device__ float myatomicAdd(float* address, float val)
// {
//     unsigned long long int* address_as_ull =
//                               (unsigned long long int*)address;
//     unsigned long long int old = *address_as_ull, assumed;

//     do {
//         assumed = old;
//         old = atomicCAS(address_as_ull, assumed,
//                         __float_as_longlong(val +
//                                __longlong_as_float(assumed)));

//     // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
//     } while (assumed != old);

//     return __longlong_as_float(old);
// }
// #endif

__constant__ float constTfms[15000];  // ... or some other big enough number

// Sign of a number
__inline__ __device__
int sgn_1(float val) {
    return (float(0) < val) - (val < float(0));
}

#define WARPSIZE 32

// Warp-shuffle to compute the sum across the warp very efficiently
__inline__ __device__
float warpReduceSum(float val) {
  for (int offset = WARPSIZE/2; offset > 0; offset /= 2)
    val += __shfl_down(val, offset);
  return val;
}


/// Get the (batch,row,col) indices corresponding to a given thread index (3D point index)
__device__ void getCoordinates(const int tid, const int nrows, const int ncols,
                               int &batch, int &row, int &col)
{
    // Get col id
    int id = tid;
    col = id % ncols;
    id = id / ncols;

    // Get row id
    row = id % nrows;
    id = id / nrows;

    // Get batch id
    batch = id;
}

namespace {


template <typename scalar_t>
__global__ void NTfm3D_cuda_forward_kernel(
  const float  *points,
  const float  *masks,
  float  *tfmpoints,
  int nrows, int ncols, int npoints, int nSE3,
  long ps0, long ps1, long ps2, long ps3,
  long ms0, long ms1, long ms2, long ms3,
  long ts0, long ts1, long ts2, long ts3) {

  // column index
  const int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id >= npoints) return;


  // Get the batch, row and column indices
  int b,r,c;
  getCoordinates(id, nrows, ncols, b, r, c);

  // Get 3D input point (p)
  int valp = b*ps0 + r*ps2 + c*ps3; // Don't add stride along 3D dim
  float x = *(points + 0*ps1 + valp);
  float y = *(points + 1*ps1 + valp);
  float z = *(points + 2*ps1 + valp);

  // Compute p + sum_k w_k * (R_k*p + t_k) across the different SE3s
  int valm = b*ms0 + r*ms2 + c*ms3;
  float xt = 0, yt = 0, zt = 0;
  // float xt = x, yt = y, zt = z;

  for (int k = 0; k < nSE3; k++)
  {
    // Get transform & wt
    float w_k = *(masks + k*ms1 + valm);  // Get the weight for the 'k'th transform "
    float *T = constTfms + b*ts0 + k*ts1; // Get the 'k'th transform

    // Add w_k * (R_k*p + t_k) (for X,Y,Z coordinates)
    // xt += w_k * (T[0] * x + T[1] * y + T[2]  * z + T[3] - x); // w_k * (R_k * p_x + t_k)
    // yt += w_k * (T[4] * x + T[5] * y + T[6]  * z + T[7] - y); // w_k * (R_k * p_y + t_k)
    // zt += w_k * (T[8] * x + T[9] * y + T[10] * z + T[11] - z); // w_k * (R_k * p_z + t_k)

    xt += w_k * (T[0] * x + T[1] * y + T[2]  * z + T[3]); // w_k * (R_k * p_x + t_k)
    yt += w_k * (T[4] * x + T[5] * y + T[6]  * z + T[7]); // w_k * (R_k * p_y + t_k)
    zt += w_k * (T[8] * x + T[9] * y + T[10] * z + T[11]); // w_k * (R_k * p_z + t_k)

  }

  // Copy to output
  *(tfmpoints + 0*ps1 + valp) = xt;
  *(tfmpoints + 1*ps1 + valp) = yt;
  *(tfmpoints + 2*ps1 + valp) = zt;

}


template <typename scalar_t>
__global__ void NTfm3D_cuda_backward_kernel(const float  *points,
                                            const float  *masks,
                                            float  *gradPoints,
                                            float  *gradMasks,
                                            float  *gradTfms,
                                            const float  *gradTfmpoints,
                                            int useMaskGradMag,
                                            int nrows, int ncols, int nSE3,
                                            long ps0, long ps1, long ps2, long ps3,
                                            long ms0, long ms1, long ms2, long ms3,
                                            long ts0, long ts1, long ts2, long ts3) {
  
  // Get the row, col, batch IDs & figure out if we are within limits
  int c = (blockIdx.x * blockDim.x) + threadIdx.x; // col ID (innermost dimension in our data for coalescing)
  int r = (blockIdx.y * blockDim.y) + threadIdx.y; // row ID
  int b = blockIdx.z; // Batch ID (since blockDim.z = 1, theadIdx.z = 0)
  bool withinLimits = ((c < ncols) && (r < nrows));

  // Create a shared memory buffer for storing the gradients w.r.t a single transform
  extern __shared__ float sharedData[];

  // Declare temp vars
  int tid = threadIdx.x + threadIdx.y * blockDim.x; // Id of thread in local block
  int nThreads = blockDim.x * blockDim.y;
  int nThreads2 = nThreads/2;
  int nSharedGrads  = nThreads * 12;
  int nSharedGradResults = nSE3*12;
  float *sharedGradTfms = sharedData; // nThreads*12
  float *sharedGradTfmResults = (float *)&sharedData[nSharedGrads]; // nSE3*12

  // Get 3D input point (p) & gradient w.r.t output point (gpt). Read only if inside limits
  float x, y, z, gxt, gyt, gzt;
  int valp = b*ps0 + r*ps2 + c*ps3; // Don't add stride along 3D dim
  if (withinLimits)
  {
      x = *(points + 0*ps1 + valp);
      y = *(points + 1*ps1 + valp);
      z = *(points + 2*ps1 + valp);

      // Get gradient w.r.t output point (gpt)
      gxt = *(gradTfmpoints + 0*ps1 + valp);
      gyt = *(gradTfmpoints + 1*ps1 + valp);
      gzt = *(gradTfmpoints + 2*ps1 + valp);
  }

  // Compute the gradients over all the transforms from a given 3D point
  int valm = b*ms0 + r*ms2 + c*ms3;
  float gx = 0, gy = 0, gz = 0; // Grads w.r.t input pts
  // float gxt_x = 0, gxt_y = 0, gxt_z = 0, gyt_x = 0, gyt_y = 0, gyt_z = 0, gzt_x = 0, gzt_y = 0, gzt_z = 0;
  for(int k = 0; k < nSE3; k++)
  {
      // Compute all the gradients if within limits or set the grads to zero
      if(withinLimits)
      {
          // Get transform & wt
          float w_k = *(masks + k*ms1 + valm);   // Get the weight for the 'k'th transform "
          float *T  = constTfms + b*ts0 + k*ts1; // Get the 'k'th transform

          // Create temp scalars
          float tx = (T[0] * gxt + T[4] * gyt + T[8]  * gzt);
          float ty = (T[1] * gxt + T[5] * gyt + T[9]  * gzt);
          float tz = (T[2] * gxt + T[6] * gyt + T[10] * gzt);

          // === Gradient w.r.t input point (p = R^T * gpt, summed across all the "k" transforms)
          gx += w_k * tx;
          gy += w_k * ty;
          gz += w_k * tz;

          // gxt_x += w_k * (T[0] - 1);
          // gxt_y += w_k * T[1];
          // gxt_z += w_k * T[2];

          // gyt_x += w_k * T[4];
          // gyt_y += w_k * (T[5] - 1);
          // gyt_z += w_k * T[6];

          // gzt_x += w_k * T[8];
          // gzt_y += w_k * T[9];
          // gzt_z += w_k * (T[10] - 1);



          // float gxt_wk = T[0] * x + T[1] * y + T[2]  * z + T[3] - x;
          // float gyt_wk = T[4] * x + T[5] * y + T[6]  * z + T[7] - y;
          // float gzt_wk = T[8] * x + T[9] * y + T[10] * z + T[11] - z;


          // === Gradient w.r.t mask (w_k) = (R_k^T * p + t_k) * gpt
          if (useMaskGradMag)
              *(gradMasks + k*ms1 + valm) = x * tx + y * ty + z * tz +
                                        gxt * T[3] + gyt * T[7] + gzt * T[11];

              // *(gradMasks + k*ms1 + valm) = gxt * gxt_wk + gyt * gyt_wk + gzt * gzt_wk;

          // else
          //     *(gradMasks + k*ms1 + valm) = sgn_1(gxt) * (T[0] * x + T[1] * y + T[2]  * z + T[3]) +
          //                                   sgn_1(gyt) * (T[4] * x + T[5] * y + T[6]  * z + T[7]) +
          //                                   sgn_1(gzt) * (T[8] * x + T[9] * y + T[10] * z + T[11]); // Use only sign

          // === Gradients w.r.t transforms (t_k), stored in shared memory
          // Grads w.r.t rotation parameters (sum across all pts)
          // First nThreads params is Tfm(0,0), next is Tfm(0,1) etc for removing memory bank conflicts when reading to shared memory
          sharedGradTfms[0*nThreads+tid]  = w_k * x * gxt;
          sharedGradTfms[1*nThreads+tid]  = w_k * y * gxt;
          sharedGradTfms[2*nThreads+tid]  = w_k * z * gxt;
          sharedGradTfms[4*nThreads+tid]  = w_k * x * gyt;
          sharedGradTfms[5*nThreads+tid]  = w_k * y * gyt;
          sharedGradTfms[6*nThreads+tid]  = w_k * z * gyt;
          sharedGradTfms[8*nThreads+tid]  = w_k * x * gzt;
          sharedGradTfms[9*nThreads+tid]  = w_k * y * gzt;
          sharedGradTfms[10*nThreads+tid] = w_k * z * gzt;

          // Grads w.r.t translation parameters (sum across all pts)
          sharedGradTfms[3*nThreads+tid]  = w_k * gxt;
          sharedGradTfms[7*nThreads+tid]  = w_k * gyt;
          sharedGradTfms[11*nThreads+tid] = w_k * gzt;
      }
      else
      {
          // Re-initialize shared memory to zero (no need to sync here as we don't += to this memory till we do a syncthreads later)
          for(int i = tid; i < nSharedGrads; i+=nThreads)
              sharedGradTfms[i] = 0;
      }
      __syncthreads(); // Synchronize all threads before we sum up the tfm gradients

      // === Do the parallel reduce for that particular transform dimension
      // === ASSUMPTION: We have power of 2 block sizes!
      // From: Slide 22 of http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
      // We use first half of threads to compute sums for first 6 transform params & the rest for the last 6 params
      for(unsigned int s = nThreads2; s>=32; s>>=1)
      {
          // Second nThreads/2 elements will be added to first nThreads/2 elements, then
          // Second nThreads/4 elements will be added to first nThreads/4 elements and so on!
          if (tid < s)
          {
              // Sum up gradients w.r.t first 6 parameters!
              for(int i = 0; i < 6; i++)
                  sharedGradTfms[i*nThreads + tid] += sharedGradTfms[i*nThreads + tid + s];
          }
          else if((tid >= nThreads2) && (tid - nThreads2) < s) // Use the second half of threads to process the remaining 6 transform parameters
          {
              // Sum up gradients w.r.t last 6 parameters!
              for(int i = 6; i < 12; i++)
                  sharedGradTfms[i*nThreads + tid - nThreads2] += sharedGradTfms[i*nThreads + tid - nThreads2 + s];
          }
          __syncthreads();
      }

      // This uses warp-shuffle to compute the sum across a warp (32 threads)
      // Note that for this to work, you have to have run the loop until the sum is computed for the first 32 threads in the warp
      if (tid < 32)
      {
          for(int i = 0; i < 12; i++)
          {
              float sum = warpReduceSum(sharedGradTfms[i*nThreads + tid]);
              if (tid == 0)
                  sharedGradTfmResults[k*12+i] = sum; // Store final summed result in shared memory, we can copy to global later in parallel
          }
      }
  }
  __syncthreads(); // Wait till all gradients have been propely summed up!

  // Add computed tfm gradients to global memory in parallel!
  for(int i = tid; i < nSharedGradResults; i+=nThreads)
      atomicAdd(gradTfms + b*ts0 + i, sharedGradTfmResults[i]); // Final value corresponding to that term of the tfm

  // Gradients w.r.t pts (copy after sum across tfms)
  if (withinLimits)
  {

      // gxt_x = gxt_x + 1;
      // gyt_y = gyt_y + 1;
      // gzt_z = gzt_z + 1;

      // gx = gxt * gxt_x + gyt * gyt_x + gzt * gzt_x;
      // gy = gxt * gxt_y + gyt * gyt_y + gzt * gzt_y;
      // gz = gxt * gxt_z + gyt * gyt_z + gzt * gzt_z;

      *(gradPoints + 0*ps1 + valp) = gx;
      *(gradPoints + 1*ps1 + valp) = gy;
      *(gradPoints + 2*ps1 + valp) = gz;
  }
}

} // namespace


int NTfm3D_cuda_forward(const torch::Tensor *points,
                        const torch::Tensor *masks,
                        const torch::Tensor *tfms,
                        torch::Tensor *tfmpoints) {

  auto points_accessor = points->packed_accessor<float,4>();
  auto masks_accessor = masks->packed_accessor<float,4>();
  auto tfms_accessor = tfms->packed_accessor<float,4>();

  // Initialize vars
  int batchSize = points_accessor.size(0);
  int ndim      = points_accessor.size(1);
  int nrows     = points_accessor.size(2);
  int ncols     = points_accessor.size(3);
  int nSE3      = masks_accessor.size(1);
  assert(ndim == 3); // 3D points

  // int nTfmParams = points->numel();
  int nTfmParams = tfms->numel();

    //Get data pointers
  float *points_data    = points->data<float>();
  float *masks_data     = masks->data<float>();
  float *tfms_data      = tfms->data<float>();
  float *tfmpoints_data = tfmpoints->data<float>();
  
  // Copy transforms to constant memory to reduce global memory read overhead
  hipMemcpyToSymbol(HIP_SYMBOL(constTfms), tfms_data, nTfmParams * sizeof(float));

  // Block and thread structure - we have one large set of points, so use 1d block/threads
  int npoints = batchSize * nrows * ncols;
  int numBlocks = ceil(npoints * (1.0/256));
  dim3 blocks(numBlocks);
  dim3 threads(256);

  // Get strides
  long ps[4] = {points_accessor.stride(0), points_accessor.stride(1), points_accessor.stride(2), points_accessor.stride(3)};
  long ms[4] = {masks_accessor.stride(0), masks_accessor.stride(1), masks_accessor.stride(2), masks_accessor.stride(3)};
  long ts[4] = {tfms_accessor.stride(0), tfms_accessor.stride(1), tfms_accessor.stride(2), tfms_accessor.stride(3)};




  // // Timer
  // hipEvent_t start, stop;
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord(start);

  AT_DISPATCH_FLOATING_TYPES(points->type(), "NTfm3D_forward_cuda", ([&] {
    NTfm3D_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
        // points->packed_accessor<scalar_t,4,torch::RestrictPtrTraits,size_t>(),
        // masks->packed_accessor<scalar_t,4,torch::RestrictPtrTraits,size_t>(),
        // tfms->packed_accessor<scalar_t,4,torch::RestrictPtrTraits,size_t>(),
        // tfmpoints->packed_accessor<scalar_t,4,torch::RestrictPtrTraits,size_t>(),
        points_data,
        masks_data,
        tfmpoints_data,
        nrows, ncols, npoints, nSE3,
        (int) ps[0], (int) ps[1], (int) ps[2], (int) ps[3],
        (int) ms[0], (int) ms[1], (int) ms[2], (int) ms[3],
        (int) ts[0], (int) ts[1], (int) ts[2], (int) ts[3]
        );
  }));

  // Wait for kernel to finish
  hipDeviceSynchronize();

  // Finish timing and show stats
  // hipEventRecord(stop);
  // hipEventSynchronize(stop);
  // float milliseconds = 0;
  // hipEventElapsedTime(&milliseconds, start, stop);
  // printf("FWD: Time taken in milliseconds: %f\n",milliseconds);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
      printf("error in NTfm3D_ForwardLauncher: %s\n", hipGetErrorString(err));
      assert(false);
  }

  return 1;
}

int NTfm3D_cuda_backward(const torch::Tensor *points,
                         const torch::Tensor *masks,
                         const torch::Tensor *tfms,
                         const torch::Tensor *tfmpoints,
                         torch::Tensor *gradPoints,
                         torch::Tensor *gradMasks,
                         torch::Tensor *gradTfms,
                         const torch::Tensor *gradTfmpoints,
                         int useMaskGradMag) {

  auto points_accessor = points->packed_accessor<float,4>();
  auto masks_accessor = masks->packed_accessor<float,4>();
  auto tfms_accessor = tfms->packed_accessor<float,4>();

  // Initialize vars
  int batchSize = points_accessor.size(0);
  int ndim      = points_accessor.size(1);
  int nrows     = points_accessor.size(2);
  int ncols     = points_accessor.size(3);
  int nSE3      = masks_accessor.size(1);
  assert(ndim == 3); // 3D points

  // int nTfmParams = points->numel();
  int nTfmParams = tfms->numel();


  //Get data pointers
  float *points_data    = points->data<float>();
  float *masks_data     = masks->data<float>();
  float *tfms_data      = tfms->data<float>();
  float *tfmpoints_data = tfmpoints->data<float>();
  float *gradPoints_data  = gradPoints->data<float>();
  float *gradMasks_data      = gradMasks->data<float>();
  float *gradTfms_data      = gradTfms->data<float>();
  float *gradTfmpoints_data = gradTfmpoints->data<float>();

  
  // Copy transforms to constant memory to reduce global memory read overhead
  hipMemcpyToSymbol(HIP_SYMBOL(constTfms), tfms_data, nTfmParams * sizeof(float));

  // Compute gradients w.r.t the input tfms next
  dim3 threads(16,16,1);
  dim3 blocks(ceil(ncols*(1.0/threads.x)),ceil(nrows*(1.0/threads.y)),batchSize); // all threads in a block will access same example
  int sharedMemSize = threads.x * threads.y * 3 * 4 * sizeof(float) + nSE3 * 3 * 4 * sizeof(float); // Memory for 12 vals per thread + nSE3*12 vals for storing result
  if (sharedMemSize > 32000)
  {
      printf("Shared memory size for transform gradients (%d) > 32000. Can't be stored in shared memory."
             "Please reduce number of threads per block \n", sharedMemSize);
      assert(false); // Exit
  }


  // Set gradients w.r.t pts & tfms to zero (as we add to these in a loop later)
  gradPoints->zero_();
  gradTfms->zero_();


  // Get strides
  long ps[4] = {points_accessor.stride(0), points_accessor.stride(1), points_accessor.stride(2), points_accessor.stride(3)};
  long ms[4] = {masks_accessor.stride(0), masks_accessor.stride(1), masks_accessor.stride(2), masks_accessor.stride(3)};
  long ts[4] = {tfms_accessor.stride(0), tfms_accessor.stride(1), tfms_accessor.stride(2), tfms_accessor.stride(3)};



   // Timer
   // hipEvent_t start, stop;
   // hipEventCreate(&start);
   // hipEventCreate(&stop);
   // hipEventRecord(start);

  AT_DISPATCH_FLOATING_TYPES(points->type(), "NTfm3D_backward_cuda", ([&] {
    NTfm3D_cuda_backward_kernel<scalar_t><<<blocks, threads, sharedMemSize>>>(
        points_data,
        masks_data,
        gradPoints_data,
        gradMasks_data,
        gradTfms_data,
        gradTfmpoints_data,
        useMaskGradMag,
        nrows, ncols, nSE3,
        (int) ps[0], (int) ps[1], (int) ps[2], (int) ps[3],
        (int) ms[0], (int) ms[1], (int) ms[2], (int) ms[3],
        (int) ts[0], (int) ts[1], (int) ts[2], (int) ts[3]
        );
  }));

  // Wait for kernel to finish
  hipDeviceSynchronize();

  //    // Finish timing and show stats
  //    hipEventRecord(stop);
  //    hipEventSynchronize(stop);
  //    float milliseconds = 0;
  //    hipEventElapsedTime(&milliseconds, start, stop);
  //    printf("BWD: Time taken in milliseconds: %f\n",milliseconds);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in NTfm3D_BackwardLauncher: %s\n", hipGetErrorString(err));
    assert(false);
  }


  return 1;
}
